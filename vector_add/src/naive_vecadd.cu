#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../include/vec_add_kernel.h"

__global__
void naive_vector_add(float *a , float *b , float *c , int n){
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index < n){
        c[index] = a[index] + b[index];
    }

}


void launchVecAdd(float *a, float *b, float *c , int size){
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;

    naive_vector_add<<<gridSize , blockSize>>>(a , b , c , size);
    hipDeviceSynchronize();
}