#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include "../include/relu_kernel.h"

#define CUDA_CHECK(err) \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    }


int main(){

    int N = 100000;

    int iter = 100;

    size_t mem_size = N * sizeof(float);

    float *h_input = new float[N];
    for(size_t i = 0 ; i < N ; i++){
        h_input[i] = static_cast<float>(rand()) / RAND_MAX;
    }



    float *d_input , *d_output;

    CUDA_CHECK(hipMalloc(&d_input , mem_size));
    CUDA_CHECK(hipMalloc(&d_output , mem_size));

    CUDA_CHECK(hipMemcpy(d_input, h_input, mem_size, hipMemcpyHostToDevice));

    auto benchmark_kernel = [=](void (*kernel)(float * , float * , int) , const char* name){
        kernel(d_input , d_output , N);
        

        hipEvent_t start , stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);


        CUDA_CHECK(hipEventRecord(start));
        for(int i = 0 ; i < iter ; i++){
            kernel(d_input , d_output , N);
        }
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float elapsed_ms;
        CUDA_CHECK(hipEventElapsedTime(&elapsed_ms, start, stop));
        std::cout << name << " average time: " << (elapsed_ms / iter) << " ms" << std::endl;
        CUDA_CHECK(hipEventDestroy(start));
        CUDA_CHECK(hipEventDestroy(stop));
    };

    benchmark_kernel(launch_relu_naive,"naive ReLU");
    benchmark_kernel(launch_relu_vectorized,"Vecotrized ReLU");
    benchmark_kernel(launch_relu_optimized,"Optimized ReLU");

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
    delete[] h_input;
}