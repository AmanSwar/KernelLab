#include <hip/hip_runtime.h>
#include <pthread.h>
#include <stdio.h>
#include "../include/vec_add_kernel.h"

int main() {
    const int size = 10000000;
    float *a = new float[size];
    float *b = new float[size];
    float *c = new float[size];

    for(int i = 0; i < size; i++) {
        a[i] = static_cast<float>(i);
        b[i] = static_cast<float>(i);
    }

    float *da, *db, *dc;
    int size_allocated = size * sizeof(float);
    hipMalloc(&da, size_allocated);
    hipMalloc(&db, size_allocated);
    hipMalloc(&dc, size_allocated);

    hipMemcpy(da, a, size_allocated, hipMemcpyHostToDevice);
    hipMemcpy(db, b, size_allocated, hipMemcpyHostToDevice);
    
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    

    //for naive implementation
    hipEventRecord(start);
    launchVecAdd(da, db, dc, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float naive_time = 0;
    hipEventElapsedTime(&naive_time, start, stop);
    printf("CUDA kernel execution time (naive) : %.6f s\n", naive_time/1000);
    

    //for shared memory 
    hipEventRecord(start);
    launchShared(da, db, dc, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float shared_time = 0;
    hipEventElapsedTime(&shared_time, start, stop);
    printf("CUDA kernel execution time (shared) : %.6f s\n", shared_time/1000);
    


    
    //for coalesced memory
    hipEventRecord(start);
    launchCoalesced(da, db, dc, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float coalesced_time = 0;
    hipEventElapsedTime(&coalesced_time, start, stop);
    printf("CUDA kernel execution time (coalesced) : %.6f s\n", coalesced_time/1000);
    

    //tiling
    hipEventRecord(start);
    launchTiled(da , db ,dc , size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float tiled_time = 0;
    hipEventElapsedTime(&tiled_time, start, stop);
    printf("CUDA kernel execution time (tiled) : %.6f s\n", tiled_time/1000);
    

    //multi element processing
    hipEventRecord(start);
    launchMultiElement(da, db , dc , size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float multi_time = 0;
    hipEventElapsedTime(&multi_time, start, stop);
    printf("CUDA kernel execution time (multi) : %.6f s\n", multi_time/1000);


    hipEventRecord(start);
    launchVectorized(da, db , dc , size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float vectorized_time = 0;
    hipEventElapsedTime(&vectorized_time, start, stop);
    printf("CUDA kernel execution time (vectorized) : %.6f s\n", vectorized_time/1000);

    
    printf("\n");
    printf("Boost from shared : %2f \n" , naive_time/shared_time );
    printf("Boost from coalescing: %2f \n" , naive_time/coalesced_time );
    printf("Boost from tiled: %2f \n" , naive_time/tiled_time );
    printf("Boost from multi: %2f \n" , naive_time/multi_time );
    printf("Boost from vectorization: %2f \n" , naive_time/vectorized_time );
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipMemcpy(c, dc, size_allocated, hipMemcpyDeviceToHost);

    
    delete[] a;
    delete[] b;
    delete[] c;
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    
    return 0;
}