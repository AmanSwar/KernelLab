#include <__clang_cuda_builtin_vars.h>
#include <hip/hip_runtime.h>
#define BLOCKSIZE 16


__global__ void greyscale_shared(
    const float *input,
    float *output,
    int width,
    int height,
    int channels
){
    __shared__ float s_data[BLOCKSIZE][BLOCKSIZE][3];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;


    if(x < width && y < height){
        int pos = (y * width + x) * channels;

        s_data[ty][tx][0] = input[pos];
        s_data[ty][tx][1] = input[pos + 1];
        s_data[ty][tx][2] = input[pos + 2];

        __syncthreads();

        float r = s_data[ty][tx][0];
        float g = s_data[ty][tx][1];
        float b = s_data[ty][tx][2];


        output[y * width + x] =  0.299f * r + 0.587f * g + 0.114f * b;

    }
}


void launch_shared(const float *input , float *output , int width , int height , int channels){
    dim3 blockSize(16 , 16);
    dim3 gridSize((width + blockSize.x -1 / blockSize.x) , (height + blockSize.y - 1) / blockSize.y);
    greyscale_shared<<<gridSize , blockSize>>>(input, output, width, height, channels);
    hipDeviceSynchronize();
}