#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include "../include/reduction_kernels.h"

#define CUDA_CHECK(err) \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    }


int main(){

    int N = 100000;

    int iter = 100;

    size_t mem_size = N * sizeof(float);

    float *h_input = new float[N];
    for(size_t i = 0 ; i < N ; i++){
        h_input[i] = static_cast<float>(rand()) / RAND_MAX;
    }



    float *d_input , *d_output;

    CUDA_CHECK(hipMalloc(&d_input , mem_size));
    CUDA_CHECK(hipMalloc(&d_output , mem_size));

    CUDA_CHECK(hipMemcpy(d_input, h_input, mem_size, hipMemcpyHostToDevice));

    auto benchmark_kernel = [=](void (*kernel)(float * , float * , int) , const char* name){
        kernel(d_input , d_output , N);
        

        hipEvent_t start , stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);


        CUDA_CHECK(hipEventRecord(start));
        for(int i = 0 ; i < iter ; i++){
            kernel(d_input , d_output , N);
        }
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float elapsed_ms;
        CUDA_CHECK(hipEventElapsedTime(&elapsed_ms, start, stop));
        std::cout << name << " average time: " << (elapsed_ms / iter) << " ms" << std::endl;
        CUDA_CHECK(hipEventDestroy(start));
        CUDA_CHECK(hipEventDestroy(stop));
    };

    benchmark_kernel(launch_naive_reduction,"naive reduction");
    benchmark_kernel(launch_no_divergence_reduction,"no divergence reduction");
    benchmark_kernel(launch_warp_optimized_reduction,"warp reduction");

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
    delete[] h_input;
}