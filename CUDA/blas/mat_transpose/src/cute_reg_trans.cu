#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cute/config.hpp>
#include <cute/layout.hpp>
#include <cute/pointer.hpp>
#include <cute/pointer_flagged.hpp>
#include <cute/stride.hpp>
#include <cute/tensor.hpp>
#include <cute/tensor_impl.hpp>


using namespace cute;

#define UNIT_BLK_SIZE 16

template <typename T , int BLK_M , int BLK_N , typename ThreadLayoutA, typename ThreadLayoutB>
__global__ void mat_trans_cute_reg_kernel(
    const T* pA , T* pB , int M , int N , ThreadLayoutA tA , ThreadLayoutB tB
){

    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    auto ma = make_tensor(make_gmem_ptr(pA) , make_layout(make_shape(M,N), GenRowMajor{}));
    auto mb = make_tensor(make_gmem_ptr(pB) , make_layout(make_shape(N , M) , GenRowMajor{}));


    auto ga = local_tile(ma , make_shape(Int<BLK_M>{} , Int<BLK_N>{}) , make_coord(bx , by));
    auto gb = local_tile(mb , make_shape(Int<BLK_N>{} , Int<BLK_M>{}) , make_coord(by , bx));

    auto ca = local_tile(make_identity_tensor(ma.shape()) , make_shape(Int<BLK_M>{} , Int<BLK_N>{}) , make_coord(bx , by));


    Tensor tAga = local_partition(ga , tA , tx);
    Tensor tBgb = local_partition(gb , tB , tx); 

    Tensor tAcA = local_partition(ca , tA , tx);

    //for bound checks -> represents a block
    Tensor tApA = make_tensor<bool>(tAcA.shape() , tAcA.stride());

    CUTE_UNROLL
    for(int i = 0 ; i < size<0>(tApA) ; i++){
        CUTE_UNROLL
        for(int j = 0 ; j < size<1>(tApA) ; j++){
            tApA(i , j) = get<0>(tAcA(i,j)) < M && get<1>(tAcA(i,j)) < N;
        }
    }

    copy_if(tApA , tAga , tBgb);
}

// void mat_transpose_cute_row2col_reg(torch::Tensor x, torch::Tensor y) {
void mat_transpose_cute_row2col_reg(float*  x, float* y , int M , int N) {

  const int BM = UNIT_BLK_SIZE;
  const int BN = UNIT_BLK_SIZE;
  
  
  auto tA = make_layout(make_shape(Int<BM>{}, Int<BN>{}), GenColMajor{});
  auto tB = make_layout(make_shape(Int<BN>{}, Int<BM>{}), GenRowMajor{});
  
  static_assert(size(tA) == size(tB));
  
  dim3 block(size(tA));
  dim3 grid((M + BM - 1) / BM, (N + BN - 1) / BN);

  mat_trans_cute_reg_kernel<float, BM, BN, decltype(tA), decltype(tB)>
      <<<grid, block>>>(x, y, M, N, tA, tB);
    //   <<<grid, block>>>(x.data_ptr<float>(), y.data_ptr<float>(), M, N, tA, tB);
}



// ======================================================
#define N 2048 // Matrix dimensions (N x N)

// Helper to fill the matrix with sequential values.
void fillMatrix(float *mat, int n) {
  for (int i = 0; i < n * n; ++i) {
    mat[i] = static_cast<float>(i);
  }
}

// Verify that matrix B is the transpose of matrix A.
bool verifyTranspose(const float *A, const float *B, int n) {
  for (int i = 0; i < n; ++i) {
    for (int j = 0; j < n; ++j) {
      if (A[i * n + j] != B[j * n + i]) {
        return false;
      }
    }
  }
  return true;
}

int main() {
  // Calculate the size in bytes of the matrix.
  int size = N * N;
  size_t bytes = size * sizeof(float);

  // Allocate host memory.
  float *h_A = (float *)malloc(bytes);
  float *h_B = (float *)malloc(bytes);

  // Initialize the input matrix.
  fillMatrix(h_A, N);

  // Allocate device memory.
  float *d_A, *d_B;
  hipMalloc(&d_A, bytes);
  hipMalloc(&d_B, bytes);

  // Copy the host matrix to device.
  hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);

  // Create CUDA events for timing.
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // ---- Benchmark the Naïve Transpose Kernel ----
  hipEventRecord(start);
  for(int i = 0; i < 100; i++){
    mat_transpose_cute_row2col_reg(d_A, d_B, N , N);
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float ms_naive = 0;
  hipEventElapsedTime(&ms_naive, start, stop);

  // Copy the result back to host and verify correctness.
  hipMemcpy(h_B, d_B, bytes, hipMemcpyDeviceToHost);
  if (verifyTranspose(h_A, h_B, N)) {
    std::cout << "Naïve kernel: Transpose is correct." << std::endl;
  } else {
    std::cout << "Naïve kernel: Transpose is incorrect!" << std::endl;
  }
  std::cout << "Naïve kernel execution time: " << ms_naive/100 << " ms"
            << std::endl;
  hipFree(d_A);
  hipFree(d_B);
  free(h_A);
  free(h_B);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
